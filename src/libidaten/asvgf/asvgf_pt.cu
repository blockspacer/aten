#include "hip/hip_runtime.h"
#include "asvgf/asvgf.h"

#include "kernel/StreamCompaction.h"

#include "kernel/context.cuh"
#include "kernel/light.cuh"
#include "kernel/material.cuh"
#include "kernel/intersect.cuh"
#include "kernel/accelerator.cuh"
#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

__global__ void genPathASVGF(
    idaten::TileDomain tileDomain,
    bool isFillAOV,
    idaten::SVGFPathTracing::Path* paths,
    aten::ray* rays,
    int width, int height,
    int maxBounces,
    unsigned int frame,
    const aten::CameraParameter* __restrict__ camera,
    hipTextureObject_t blueNoise,
    int blueNoiseResW, int blueNoiseResH, int blueNoiseLayerNum,
    const unsigned int* __restrict__ random)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    const auto idx = getIdx(ix, iy, width);

    paths->attrib[idx].isHit = false;

    if (paths->attrib[idx].isKill) {
        paths->attrib[idx].isTerminate = true;
        return;
    }

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths->sampler[idx].init(frame, 0, scramble, samplerValues);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f * ((frame + 133 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths->sampler[idx].init(frame % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM), 0, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_BLUENOISE
    paths->sampler[idx].init(
        ix, iy, frame,
        maxBounces,
        idaten::SVGFPathTracing::ShadowRayNum,
        blueNoiseResW, blueNoiseResH, blueNoiseLayerNum,
        blueNoise);
#endif

    float r1 = paths->sampler[idx].nextSample();
    float r2 = paths->sampler[idx].nextSample();

    if (isFillAOV) {
        r1 = r2 = 0.5f;
    }

    ix += tileDomain.x;
    iy += tileDomain.y;

    float s = (ix + r1) / (float)(camera->width);
    float t = (iy + r2) / (float)(camera->height);

    AT_NAME::CameraSampleResult camsample;
    AT_NAME::PinholeCamera::sample(&camsample, camera, s, t);

    rays[idx] = camsample.r;

    paths->throughput[idx].throughput = aten::vec3(1);
    paths->throughput[idx].pdfb = 0.0f;
    paths->attrib[idx].isTerminate = false;
    paths->attrib[idx].isSingular = false;

    paths->contrib[idx].samples += 1;

    // Accumulate value, so do not reset.
    //path.contrib = aten::vec3(0);
}

__global__ void shadeASVGF(
    idaten::TileDomain tileDomain,
    float4* aovNormalDepth,
    float4* aovTexclrMeshid,
    aten::mat4 mtxW2C,
    int width, int height,
    idaten::SVGFPathTracing::Path* paths,
    const int* __restrict__ hitindices,
    int* hitnum,
    const aten::Intersection* __restrict__ isects,
    aten::ray* rays,
    int frame,
    int bounce, int rrBounce,
    const aten::GeomParameter* __restrict__ shapes, int geomnum,
    const aten::MaterialParameter* __restrict__ mtrls,
    const aten::LightParameter* __restrict__ lights, int lightnum,
    const aten::PrimitiveParamter* __restrict__ prims,
    hipTextureObject_t vtxPos,
    hipTextureObject_t vtxNml,
    const aten::mat4* __restrict__ matrices,
    hipTextureObject_t* textures,
    unsigned int* random,
    hipTextureObject_t blueNoise,
    idaten::SVGFPathTracing::ShadowRay* shadowRays)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx >= *hitnum) {
        return;
    }

    Context ctxt;
    {
        ctxt.geomnum = geomnum;
        ctxt.shapes = shapes;
        ctxt.mtrls = mtrls;
        ctxt.lightnum = lightnum;
        ctxt.lights = lights;
        ctxt.prims = prims;
        ctxt.vtxPos = vtxPos;
        ctxt.vtxNml = vtxNml;
        ctxt.matrices = matrices;
        ctxt.textures = textures;
    }

    idx = hitindices[idx];

    __shared__ idaten::SVGFPathTracing::ShadowRay shShadowRays[64 * idaten::SVGFPathTracing::ShadowRayNum];
    __shared__ aten::MaterialParameter shMtrls[64];

    const auto ray = rays[idx];

#if IDATEN_SAMPLER == IDATEN_SAMPLER_SOBOL
    auto scramble = random[idx] * 0x1fe3434f;
    paths->sampler[idx].init(frame, 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_CMJ
    auto rnd = random[idx];
    auto scramble = rnd * 0x1fe3434f * ((frame + 331 * rnd) / (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM));
    paths->sampler[idx].init(frame % (aten::CMJ::CMJ_DIM * aten::CMJ::CMJ_DIM), 4 + bounce * 300, scramble);
#elif IDATEN_SAMPLER == IDATEN_SAMPLER_BLUENOISE
    // Not need to do.
#endif

    aten::hitrecord rec;

    const auto& isect = isects[idx];

    auto obj = &ctxt.shapes[isect.objid];
    evalHitResult(&ctxt, obj, ray, &rec, &isect);

    bool isBackfacing = dot(rec.normal, -ray.dir) < 0.0f;

    // �����ʒu�̖@��.
    // ���̂���̃��C�̓��o���l��.
    aten::vec3 orienting_normal = rec.normal;

    if (rec.mtrlid >= 0) {
        shMtrls[threadIdx.x] = ctxt.mtrls[rec.mtrlid];

#if 1
        if (rec.isVoxel)
        {
            // Replace to lambert.
            const auto& albedo = ctxt.mtrls[rec.mtrlid].baseColor;
            shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
            shMtrls[threadIdx.x].baseColor = albedo;
        }
#endif

        if (shMtrls[threadIdx.x].type != aten::MaterialType::Layer) {
            shMtrls[threadIdx.x].albedoMap = (int)(shMtrls[threadIdx.x].albedoMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].albedoMap] : -1);
            shMtrls[threadIdx.x].normalMap = (int)(shMtrls[threadIdx.x].normalMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].normalMap] : -1);
            shMtrls[threadIdx.x].roughnessMap = (int)(shMtrls[threadIdx.x].roughnessMap >= 0 ? ctxt.textures[shMtrls[threadIdx.x].roughnessMap] : -1);
        }
    }
    else {
        // TODO
        shMtrls[threadIdx.x] = aten::MaterialParameter(aten::MaterialType::Lambert, MaterialAttributeLambert);
        shMtrls[threadIdx.x].baseColor = aten::vec3(1.0f);
    }


    // Render AOVs.
    // NOTE
    // �����ɖ@����AOV�ɕێ�����Ȃ�A�@���}�b�v�K�p�シ��ׂ�.
    // �������Atemporal reprojection�Aatrous�Ȃǂ̃t�B���^�K�p���ɖ@�����Q�Ƃ���ۂɁA�@���}�b�v���ׂ������Ă͂�����Ă��܂����Ƃ�����.
    // ����ɂ��A�t�B���^�����������悤�ɂ����炸�t�B���^�̕i�����������Ă��܂���肪��������.
    if (bounce == 0) {
        int ix = idx % tileDomain.w;
        int iy = idx / tileDomain.w;

        ix += tileDomain.x;
        iy += tileDomain.y;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        // normal, depth
        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);

        // texture color, meshid.
        auto texcolor = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1.0f));
#if 0
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.meshid);
#else
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.mtrlid);
#endif

        // For exporting separated albedo.
        shMtrls[threadIdx.x].albedoMap = -1;
    }
    // TODO
    // How to deal Refraction?
    else if (bounce == 1 && paths->attrib[idx].mtrlType == aten::MaterialType::Specular) {
        int ix = idx % tileDomain.w;
        int iy = idx / tileDomain.w;

        ix += tileDomain.x;
        iy += tileDomain.y;

        const auto _idx = getIdx(ix, iy, width);

        // World coordinate to Clip coordinate.
        aten::vec4 pos = aten::vec4(rec.p, 1);
        pos = mtxW2C.apply(pos);

        // normal, depth
        aovNormalDepth[_idx] = make_float4(orienting_normal.x, orienting_normal.y, orienting_normal.z, pos.w);

        // texture color.
        auto texcolor = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1.0f));
#if 0
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.meshid);
#else
        aovTexclrMeshid[_idx] = make_float4(texcolor.x, texcolor.y, texcolor.z, isect.mtrlid);
#endif

        // For exporting separated albedo.
        shMtrls[threadIdx.x].albedoMap = -1;
    }

    // Implicit conection to light.
    if (shMtrls[threadIdx.x].attrib.isEmissive) {
        if (!isBackfacing) {
            float weight = 1.0f;

            if (bounce > 0 && !paths->attrib[idx].isSingular) {
                auto cosLight = dot(orienting_normal, -ray.dir);
                auto dist2 = aten::squared_length(rec.p - ray.org);

                if (cosLight >= 0) {
                    auto pdfLight = 1 / rec.area;

                    // Convert pdf area to sradian.
                    // http://www.slideshare.net/h013/edubpt-v100
                    // p31 - p35
                    pdfLight = pdfLight * dist2 / cosLight;

                    weight = paths->throughput[idx].pdfb / (pdfLight + paths->throughput[idx].pdfb);
                }
            }

            auto contrib = paths->throughput[idx].throughput * weight * shMtrls[threadIdx.x].baseColor;
            paths->contrib[idx].contrib += make_float3(contrib.x, contrib.y, contrib.z);
        }

        // When ray hit the light, tracing will finish.
        paths->attrib[idx].isTerminate = true;
        return;
    }

    if (!shMtrls[threadIdx.x].attrib.isTranslucent && isBackfacing) {
        orienting_normal = -orienting_normal;
    }

    // Apply normal map.
    int normalMap = shMtrls[threadIdx.x].normalMap;
    if (shMtrls[threadIdx.x].type == aten::MaterialType::Layer) {
        // �ŕ\�w�� NormalMap ��K�p.
        auto* topmtrl = &ctxt.mtrls[shMtrls[threadIdx.x].layer[0]];
        normalMap = (int)(topmtrl->normalMap >= 0 ? ctxt.textures[topmtrl->normalMap] : -1);
    }
    AT_NAME::applyNormalMap(normalMap, orienting_normal, orienting_normal, rec.u, rec.v);

    auto albedo = AT_NAME::sampleTexture(shMtrls[threadIdx.x].albedoMap, rec.u, rec.v, aten::vec3(1), bounce);

#if 1
#pragma unroll
    for (int i = 0; i < idaten::SVGFPathTracing::ShadowRayNum; i++) {
        shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].isActive = false;
    }

    // Explicit conection to light.
    if (!(shMtrls[threadIdx.x].attrib.isSingular || shMtrls[threadIdx.x].attrib.isTranslucent))
    {
        auto shadowRayOrg = rec.p + AT_MATH_EPSILON * orienting_normal;

        for (int i = 0; i < idaten::SVGFPathTracing::ShadowRayNum; i++) {
            real lightSelectPdf = 1;
            aten::LightSampleResult sampleres;

            // TODO
            // Importance sampling.
            int lightidx = aten::cmpMin<int>(paths->sampler[idx].nextSample() * lightnum, lightnum - 1);
            lightSelectPdf = 1.0f / lightnum;

            aten::LightParameter light;
            light.pos = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 0];
            light.dir = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 1];
            light.le = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 2];
            light.v0 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 3];
            light.v1 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 4];
            light.v2 = ((aten::vec4*)ctxt.lights)[lightidx * aten::LightParameter_float4_size + 5];
            //auto light = ctxt.lights[lightidx];

            sampleLight(&sampleres, &ctxt, &light, rec.p, orienting_normal, &paths->sampler[idx], bounce);

            const auto& posLight = sampleres.pos;
            const auto& nmlLight = sampleres.nml;
            real pdfLight = sampleres.pdf;

            auto dirToLight = normalize(sampleres.dir);
            auto distToLight = length(posLight - rec.p);

            auto tmp = rec.p + dirToLight - shadowRayOrg;
            auto shadowRayDir = normalize(tmp);

            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].isActive = true;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].rayorg = shadowRayOrg;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].raydir = shadowRayDir;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].targetLightId = lightidx;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].distToLight = distToLight;
            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].lightcontrib = aten::vec3(0);
            {
                auto cosShadow = dot(orienting_normal, dirToLight);

                real pdfb = samplePDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v);
                auto bsdf = sampleBSDF(&ctxt, &shMtrls[threadIdx.x], orienting_normal, ray.dir, dirToLight, rec.u, rec.v, albedo);

                bsdf *= paths->throughput[idx].throughput;

                // Get light color.
                auto emit = sampleres.finalColor;

                if (light.attrib.isSingular || light.attrib.isInfinite) {
                    if (pdfLight > real(0) && cosShadow >= 0) {
                        // TODO
                        // �W�I���g���^�[���̈����ɂ���.
                        // singular light �̏ꍇ�́AfinalColor �ɋ����̏��Z���܂܂�Ă���.
                        // inifinite light �̏ꍇ�́A���������ɂȂ�ApdfLight�Ɋ܂܂�鋗�������Ƒł����������H.
                        // �i�ł����������̂ŁApdfLight�ɂ͋��������͊܂�ł��Ȃ��j.
                        auto misW = pdfLight / (pdfb + pdfLight);

                        shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].lightcontrib =
                            (misW * bsdf * emit * cosShadow / pdfLight) / lightSelectPdf / (float)idaten::SVGFPathTracing::ShadowRayNum;
                    }
                }
                else {
                    auto cosLight = dot(nmlLight, -dirToLight);

                    if (cosShadow >= 0 && cosLight >= 0) {
                        auto dist2 = aten::squared_length(sampleres.dir);
                        auto G = cosShadow * cosLight / dist2;

                        if (pdfb > real(0) && pdfLight > real(0)) {
                            // Convert pdf from steradian to area.
                            // http://www.slideshare.net/h013/edubpt-v100
                            // p31 - p35
                            pdfb = pdfb * cosLight / dist2;

                            auto misW = pdfLight / (pdfb + pdfLight);

                            shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i].lightcontrib =
                                (misW * (bsdf * emit * G) / pdfLight) / lightSelectPdf / (float)idaten::SVGFPathTracing::ShadowRayNum;;
                        }
                    }
                }
            }
        }
    }
#endif

    real russianProb = real(1);

    if (bounce > rrBounce) {
        auto t = normalize(paths->throughput[idx].throughput);
        auto p = aten::cmpMax(t.r, aten::cmpMax(t.g, t.b));

        russianProb = paths->sampler[idx].nextSample();

        if (russianProb >= p) {
            //shPaths[threadIdx.x].contrib = aten::vec3(0);
            paths->attrib[idx].isTerminate = true;
        }
        else {
            russianProb = max(p, 0.01f);
        }
    }
            
    AT_NAME::MaterialSampling sampling;

    sampleMaterial(
        &sampling,
        &ctxt,
        &shMtrls[threadIdx.x],
        orienting_normal,
        ray.dir,
        rec.normal,
        &paths->sampler[idx],
        rec.u, rec.v,
        albedo);

    auto nextDir = normalize(sampling.dir);
    auto pdfb = sampling.pdf;
    auto bsdf = sampling.bsdf;

    real c = 1;
    if (!shMtrls[threadIdx.x].attrib.isSingular) {
        // TODO
        // AMD�̂�abs���Ă��邪....
        c = aten::abs(dot(orienting_normal, nextDir));
        //c = dot(orienting_normal, nextDir);
    }

    if (pdfb > 0 && c > 0) {
        paths->throughput[idx].throughput *= bsdf * c / pdfb;
        paths->throughput[idx].throughput /= russianProb;
    }
    else {
        paths->attrib[idx].isTerminate = true;
    }

    // Make next ray.
    rays[idx] = aten::ray(rec.p, nextDir);

    paths->throughput[idx].pdfb = pdfb;
    paths->attrib[idx].isSingular = shMtrls[threadIdx.x].attrib.isSingular;
    paths->attrib[idx].mtrlType = shMtrls[threadIdx.x].type;

#pragma unroll
    for (int i = 0; i < idaten::SVGFPathTracing::ShadowRayNum; i++) {
        shadowRays[idx * idaten::SVGFPathTracing::ShadowRayNum + i] = shShadowRays[threadIdx.x * idaten::SVGFPathTracing::ShadowRayNum + i];
    }
}

__global__ void gatherASVGF(
    idaten::TileDomain tileDomain,
    hipSurfaceObject_t dst,
    float4* aovColorVariance,
    float4* aovMomentTemporalWeight,
    const idaten::SVGFPathTracing::Path* __restrict__ paths,
    float4* contribs,
    int width, int height)
{
    auto ix = blockIdx.x * blockDim.x + threadIdx.x;
    auto iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= tileDomain.w || iy >= tileDomain.h) {
        return;
    }

    auto idx = getIdx(ix, iy, tileDomain.w);

    auto r = paths->sampler[idx].nextSample();

    if (dst) {
        surf2Dwrite(
            make_float4(r, r, r, 1),
            dst,
            ix * sizeof(float4), iy,
            hipBoundaryModeTrap);
    }
}

namespace idaten
{
    void AdvancedSVGFPathTracing::onGenPath(
        int maxBounce,
        int seed,
        hipTextureObject_t texVtxPos,
        hipTextureObject_t texVtxNml)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        bool isFillAOV = m_mode == Mode::AOVar;

        auto blueNoise = m_bluenoise.bind();
        auto blueNoiseResW = m_bluenoise.getWidth();
        auto blueNoiseResH = m_bluenoise.getHeight();
        auto blueNoiseLayerNum = m_bluenoise.getLayerNum();

        genPathASVGF << <grid, block, 0, m_stream >> > (
            m_tileDomain,
            isFillAOV,
            m_paths.ptr(),
            m_rays.ptr(),
            m_tileDomain.w, m_tileDomain.h,
            maxBounce,
            m_frame,
            m_cam.ptr(),
            blueNoise,
            blueNoiseResW, blueNoiseResH, blueNoiseLayerNum,
            m_random.ptr());

        checkCudaKernel(genPath);
    }

    void AdvancedSVGFPathTracing::onShade(
        hipSurfaceObject_t outputSurf,
        int width, int height,
        int bounce, int rrBounce,
        hipTextureObject_t texVtxPos,
        hipTextureObject_t texVtxNml)
    {
        m_mtxW2V.lookat(
            m_camParam.origin,
            m_camParam.center,
            m_camParam.up);

        m_mtxV2C.perspective(
            m_camParam.znear,
            m_camParam.zfar,
            m_camParam.vfov,
            m_camParam.aspect);

        m_mtxC2V = m_mtxV2C;
        m_mtxC2V.invert();

        m_mtxV2W = m_mtxW2V;
        m_mtxV2W.invert();

        aten::mat4 mtxW2C = m_mtxV2C * m_mtxW2V;

        dim3 blockPerGrid(((m_tileDomain.w * m_tileDomain.h) + 64 - 1) / 64);
        dim3 threadPerBlock(64);

        auto& hitcount = m_compaction.getCount();

        int curaov = getCurAovs();

        auto blueNoise = m_bluenoise.bind();

        shadeASVGF << <blockPerGrid, threadPerBlock, 0, m_stream >> > (
            m_tileDomain,
            m_aovNormalDepth[curaov].ptr(),
            m_aovTexclrMeshid[curaov].ptr(),
            mtxW2C,
            width, height,
            m_paths.ptr(),
            m_hitidx.ptr(), hitcount.ptr(),
            m_isects.ptr(),
            m_rays.ptr(),
            m_frame,
            bounce, rrBounce,
            m_shapeparam.ptr(), m_shapeparam.num(),
            m_mtrlparam.ptr(),
            m_lightparam.ptr(), m_lightparam.num(),
            m_primparams.ptr(),
            texVtxPos, texVtxNml,
            m_mtxparams.ptr(),
            m_tex.ptr(),
            m_random.ptr(),
            blueNoise,
            m_shadowRays.ptr());

        checkCudaKernel(shade);

        onShadeByShadowRay(bounce, texVtxPos);

        m_bluenoise.unbind();
    }

    void AdvancedSVGFPathTracing::onGather(
        hipSurfaceObject_t outputSurf,
        int width, int height,
        int maxSamples)
    {
        dim3 block(BLOCK_SIZE, BLOCK_SIZE);
        dim3 grid(
            (m_tileDomain.w + block.x - 1) / block.x,
            (m_tileDomain.h + block.y - 1) / block.y);

        int curaov = getCurAovs();

        gatherASVGF << <grid, block, 0, m_stream >> > (
            m_tileDomain,
            outputSurf,
            m_aovColorVariance[curaov].ptr(),
            m_aovMomentTemporalWeight[curaov].ptr(),
            m_paths.ptr(),
            m_tmpBuf.ptr(),
            width, height);

        checkCudaKernel(gather);
    }
}
