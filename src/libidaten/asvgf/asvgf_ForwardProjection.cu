#include "hip/hip_runtime.h"
#include "asvgf/asvgf.h"

#include "kernel/pt_common.h"

#include "cuda/cudadefs.h"
#include "cuda/hip/hip_vector_types.h"
#include "cuda/cudautil.h"
#include "cuda/cudamemory.h"

#include "aten4idaten.h"

// TEA = Tiny Encryption Algorithm.
// https://en.wikipedia.org/wiki/Tiny_Encryption_Algorithm
inline __device__ void encryptTea(uint2& arg)
{
    const unsigned int key[] = {
        0xa341316c, 0xc8013ea4, 0xad90777d, 0x7e95761e
    };

    unsigned int v0 = arg.x;
    unsigned int v1 = arg.y;
    unsigned int sum = 0;
    unsigned int delta = 0x9e3779b9;

    for (int i = 0; i < 16; i++) {
        sum += delta;
        v0 += ((v1 << 4) + key[0]) ^ (v1 + sum) ^ ((v1 >> 5) + key[1]);
        v1 += ((v0 << 4) + key[2]) ^ (v0 + sum) ^ ((v0 >> 5) + key[3]);
    }

    arg.x = v0;
    arg.y = v1;
}

inline __device__ bool testReprojectedDepth(float z1, float z2, float dz)
{
    float z_diff = abs(z1 - z2);
    return z_diff < 2.0 * (dz + 1e-3f);
}

#define AT_IS_INBOUND(x, a, b)  (((a) <= (x)) && ((x) <= (b)))

__global__ void doForwardProjection(
    int4* gradientSample,
    const float4* __restrict__ curAovNormalDepth,
    const float4* __restrict__ prevAovNormalDepth,
    int frame,
    int width, int height,
    int gradientTileSize,
    float cameraDistance,
    hipSurfaceObject_t motionDetphBuffer,
    int* executedIdxArray)
{
    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;

    if (ix >= width || iy >= height) {
        return;
    }

    int idx = getIdx(ix, iy, width);

    // Compute randomized position as previous position.
    uint2 teaArg = make_uint2(idx, frame);
    encryptTea(teaArg);
    teaArg.x %= gradientTileSize;
    teaArg.y %= gradientTileSize;
    int2 idxPrev = make_int2(
        ix * gradientTileSize + teaArg.x,
        iy * gradientTileSize + teaArg.y);

    float4 motionDepth;
    surf2Dread(&motionDepth, motionDetphBuffer, idxPrev.x * sizeof(float4), idxPrev.y);

    // NOTE
    // motion = prev - cur
    //  => -motion = cur - prev
    //  => prev + (-motion) = prev + (cur - prev) = cur
    int2 idxCur = make_int2(idxPrev.x - motionDepth.x, idxPrev.y - motionDepth.y);

    // Check if idx is in screen.
    if (!AT_IS_INBOUND(idxCur.x, 0, width)
        || !AT_IS_INBOUND(idxCur.y, 0, height))
    {
        return;
    }

    int curIdx = getIdx(idxCur.x, idxCur.y, width);
    int prevIdx = getIdx(idxPrev.x, idxPrev.y, width);

    float4 curNmlDepth = curAovNormalDepth[curIdx];
    float4 prevNmlDepth = curAovNormalDepth[prevIdx];

    float pixelDistanceRatio = (curNmlDepth.w / cameraDistance) * height;

    bool accept = testReprojectedDepth(curNmlDepth.w, prevNmlDepth.w, pixelDistanceRatio);
    if (!accept) {
        return;
    }

    // Remove depth.
    curNmlDepth.w = prevNmlDepth.w = 0;

    accept = (dot(curNmlDepth, prevNmlDepth) > 0.9f);
    if (!accept) {
        return;
    }

    int2 tilePos = make_int2(
        idxCur.x % gradientTileSize,
        idxCur.y % gradientTileSize);

    // NOTE
    // Atomic functions for CUDA.
    // http://www.slis.tsukuba.ac.jp/~fujisawa.makoto.fu/cgi-bin/wiki/index.php?CUDA%A5%A2%A5%C8%A5%DF%A5%C3%A5%AF%B4%D8%BF%F4

    int res = atomicCAS(&executedIdxArray[idx], -1, idx);
    if (res < 0) {
        // NOTE
        // w is not used.
        gradientSample[idx] = make_int4(tilePos.x, tilePos.y, prevIdx, 0);

        // Rng seed.

        // Mesh id.

        // Albedo.
    }
}
